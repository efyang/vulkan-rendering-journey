#include "hip/hip_runtime.h"
#include "ngp.hpp"

namespace vkr {
	NerfRenderObject::NerfRenderObject(int texWidth, int texHeight, const std::string snapshotPath, VulkanEngine& engine):
		texWidth(texWidth), texHeight(texHeight) {
    aabbMesh = std::make_shared<Mesh>();
    renderObject = std::make_shared<RenderObject>();
    // load the snapshot
    testbed.load_snapshot(snapshotPath);

    // build centered cube mesh
    bbCenter = (testbed.m_aabb.min + testbed.m_aabb.max) * 0.5;
    Eigen::Vector3f bbMin = testbed.m_aabb.min - bbCenter;
    Eigen::Vector3f bbMax = testbed.m_aabb.max - bbCenter;
    // TODO: add free for renderObject.mesh in deletion queue

    (aabbMesh->vertices).resize(8);
    (aabbMesh->vertices)[0].position = {bbMin.x(), bbMin.y(), bbMin.z()};
    (aabbMesh->vertices)[1].position = {bbMin.x(), bbMax.y(), bbMin.z()};
    (aabbMesh->vertices)[2].position = {bbMax.x(), bbMin.y(), bbMin.z()};
    (aabbMesh->vertices)[3].position = {bbMax.x(), bbMax.y(), bbMin.z()};
    (aabbMesh->vertices)[4].position = {bbMax.x(), bbMax.y(), bbMax.z()};
    (aabbMesh->vertices)[5].position = {bbMax.x(), bbMin.y(), bbMax.z()};
    (aabbMesh->vertices)[6].position = {bbMin.x(), bbMin.y(), bbMax.z()};
    (aabbMesh->vertices)[7].position = {bbMin.x(), bbMax.y(), bbMax.z()};

    (aabbMesh->vertices)[0].uv = {0., 0.};
    (aabbMesh->vertices)[1].uv = {1., 0.};
    (aabbMesh->vertices)[2].uv = {0., 1.};
    (aabbMesh->vertices)[3].uv = {1., 1.};
    (aabbMesh->vertices)[4].uv = {0., 0.};
    (aabbMesh->vertices)[5].uv = {1., 0.};
    (aabbMesh->vertices)[6].uv = {0., 1.};
    (aabbMesh->vertices)[7].uv = {1., 1.};

    for (int i = 0; i < (aabbMesh->vertices).size(); i++) {
      (aabbMesh->vertices)[i].color = {1.0, 1.0, 1.0};
    }

    aabbMesh->indices = {1,0,2,
                      2,3,1,
                      4,3,2,
                      2,5,4,
                      7,4,5,
                      5,6,7,
                      7,6,0,
                      0,1,7,
                      1,3,4,
                      4,7,1,
                      2,0,6,
                      6,5,2};

    // this address can change with new meshes which is why it breaks?
    // or maybe changes on accesses?
    // engine.upload_mesh(engine.m_meshes[snapshotPath + "aabbMesh"]);
    engine.upload_mesh(*aabbMesh);
    engine.m_meshes[snapshotPath + "aabbMesh"] = *aabbMesh;
    // renderObject.mesh = &aabbMesh;
    // renderObject.mesh = &engine.get_mesh(snapshotPath + "aabbMesh");

    // spdlog::info("vsize {}, isize {}", renderObject.mesh->vertices.size(), renderObject.mesh->indices.size());
    cameraMatrix << 1, 0, 0, 0.5,
        0, -1,  0, 0.5,
        0,  0, -1,   2;
    prevCameraMatrix = cameraMatrix;

    // create a vulkan texture
    rawVkImage = std::make_shared<ngp::VulkanTextureSurface>(Eigen::Vector2i(texWidth, texHeight), 4);
    cudaRenderBuffer = std::make_unique<ngp::CudaRenderBuffer>(rawVkImage);
    cudaRenderBuffer->resize(Eigen::Vector2i(texWidth, texHeight));

    // default to shader readable
    engine.m_loadedTextures["nerf"] = engine.createEmptyDefaultTexture(
      texWidth, texHeight,
      vk::Format::eR8G8B8A8Srgb,
      vk::ImageLayout::eShaderReadOnlyOptimal,
      vk::AccessFlagBits::eShaderRead,
      vk::ImageUsageFlagBits::eTransferDst | vk::ImageUsageFlagBits::eSampled,
      vk::PipelineStageFlagBits::eFragmentShader);
	}

  void NerfRenderObject::update(Eigen::Matrix<float, 3, 4> camera_matrix, VulkanEngine& engine) {
    // AKA clear the frame
    cudaRenderBuffer->reset_accumulation();
    vk::Image ngpRawImage(rawVkImage->vk_image());

    // render nerf to the rawvktexture
    // Eigen::Affine3f centerShift(-bbCenter);
    // camera_matrix = camera_matrix * centerShift.matrix();
    Eigen::Vector4f rolling_shutter = Eigen::Vector4f::Zero();
    spdlog::info("Rendering nerf frame... ");
    testbed.m_fov_axis=1;
    // testbed.m_zoom=1.f;
	  testbed.m_screen_center = Eigen::Vector2f::Constant(0.5f);
    // testbed.m_scale=1.f;
    testbed.set_fov(70);
    testbed.render_frame(camera_matrix,
      camera_matrix,
      rolling_shutter,
      *cudaRenderBuffer);
    spdlog::info("Finished rendering nerf frame.");

    engine.immediate_submit([&](vk::CommandBuffer cmd) {
      vk::ImageSubresourceRange range(vk::ImageAspectFlagBits::eColor, 0, 1, 0, 1);
      // transfer texture back to transfer dst
      vk::ImageMemoryBarrier imageBarrier_toTransfer;
      imageBarrier_toTransfer.setOldLayout(vk::ImageLayout::eUndefined);
      imageBarrier_toTransfer.setNewLayout(vk::ImageLayout::eTransferDstOptimal);
      imageBarrier_toTransfer.setImage(engine.m_loadedTextures["nerf"]->image.image);
      imageBarrier_toTransfer.setSubresourceRange(range);
      // equivalent to {}
      imageBarrier_toTransfer.setSrcAccessMask(vk::AccessFlagBits::eNone);
      imageBarrier_toTransfer.setDstAccessMask(
          vk::AccessFlagBits::eTransferWrite);
      cmd.pipelineBarrier(vk::PipelineStageFlagBits::eTopOfPipe,
                          vk::PipelineStageFlagBits::eTransfer, {}, nullptr,
                          nullptr, imageBarrier_toTransfer);

      // blit from raw to render texture
      vk::ImageBlit blitRegion;
      blitRegion.srcSubresource.aspectMask = vk::ImageAspectFlagBits::eColor;
      blitRegion.srcSubresource.mipLevel = 0;
      blitRegion.srcSubresource.baseArrayLayer = 0;
      blitRegion.srcSubresource.layerCount = 1;
      blitRegion.srcOffsets[0].x = 0;
      blitRegion.srcOffsets[0].y = 0;
      blitRegion.srcOffsets[0].z = 0;
      blitRegion.srcOffsets[1].x = texWidth;
      blitRegion.srcOffsets[1].y = texHeight;
      blitRegion.srcOffsets[1].z = 1;
      blitRegion.dstSubresource.aspectMask = vk::ImageAspectFlagBits::eColor;
      blitRegion.dstSubresource.mipLevel = 0;
      blitRegion.dstSubresource.baseArrayLayer = 0;
      blitRegion.dstSubresource.layerCount = 1;
      blitRegion.dstOffsets[0] = blitRegion.srcOffsets[0];
      blitRegion.dstOffsets[1] = blitRegion.srcOffsets[1];

      cmd.blitImage(ngpRawImage,
                    vk::ImageLayout::eGeneral,
                    engine.m_loadedTextures["nerf"]->image.image,
                    vk::ImageLayout::eTransferDstOptimal,
                    blitRegion,
                    vk::Filter::eNearest);
      // now transform image back to shader optimal reading
      vk::ImageMemoryBarrier imageBarrier_toReadable;
      imageBarrier_toReadable.setImage(engine.m_loadedTextures["nerf"]->image.image);
      imageBarrier_toReadable.setOldLayout(vk::ImageLayout::eTransferDstOptimal);
      imageBarrier_toReadable.setNewLayout(
          vk::ImageLayout::eShaderReadOnlyOptimal);
      imageBarrier_toReadable.setSubresourceRange(range);
      imageBarrier_toReadable.setSrcAccessMask(
          vk::AccessFlagBits::eTransferWrite);
      imageBarrier_toReadable.setDstAccessMask(vk::AccessFlagBits::eShaderRead);
      cmd.pipelineBarrier(vk::PipelineStageFlagBits::eTransfer,
                          vk::PipelineStageFlagBits::eFragmentShader, {}, nullptr,
                          nullptr, imageBarrier_toReadable);
      spdlog::info("Converted image successfully");
    });

    prevCameraMatrix = camera_matrix;
  }


	NerfRenderObject::~NerfRenderObject() {
    spdlog::warn("nerfro destroyed!!!");
  }
}